#include "hip/hip_runtime.h"
//Udacity HW 4
//Radix Sorting

#include "utils.h"

/* Red Eye Removal
   ===============
   
   For this assignment we are implementing red eye removal.  This is
   accomplished by first creating a score for every pixel that tells us how
   likely it is to be a red eye pixel.  We have already done this for you - you
   are receiving the scores and need to sort them in ascending order so that we
   know which pixels to alter to remove the red eye.

   Note: ascending order == smallest to largest

   Each score is associated with a position, when you sort the scores, you must
   also move the positions accordingly.

   Implementing Parallel Radix Sort with CUDA
   ==========================================

   The basic idea is to construct a histogram on each pass of how many of each
   "digit" there are.   Then we scan this histogram so that we know where to put
   the output of each digit.  For example, the first 1 must come after all the
   0s so we have to know how many 0s there are to be able to start moving 1s
   into the correct position.

   1) Histogram of the number of occurrences of each digit
   2) Exclusive Prefix Sum of Histogram
   3) Determine relative offset of each digit
        For example [0 0 1 1 0 0 1]
                ->  [0 1 0 1 2 3 2]
   4) Combine the results of steps 2 & 3 to determine the final
      output location for each element and move it there

   LSB Radix sort is an out-of-place sort and you will need to ping-pong values
   between the input and output buffers we have provided.  Make sure the final
   sorted results end up in the output buffer!  Hint: You may need to do a copy
   at the end.

 */
// Take current digit of each elements
//=====================================================================
__global__ void takeCurrentDigit(unsigned int* inData, unsigned int* outDigit,
			    	 int n, int currentDigit, int nBins)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < n)
  {
    outDigit[i] = (inData[i] >> currentDigit) & (nBins - 1);
  }  
}

// Histogram on digit
//=====================================================================
__global__ void histogram(unsigned int* inData, unsigned int* hist, int n)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < n)
  {
    atomicAdd(&hist[inData[i]], 1);
  } 
}

// Inclusive scan
//=====================================================================
__global__ void inclusiveScan(unsigned int* inData, unsigned int* outData,
			      unsigned int* outLastData, int n)
{
  // 1. Each block loads data from GMEM to SMEM
  // (each thread will load 2 elements)
  extern __shared__ unsigned int s_inScanData[]; // It's size will be 2*blockDim.x (elements)
  int idx1 = (blockDim.x * blockIdx.x + threadIdx.x) * 2;
  int idx2 = (blockDim.x * blockIdx.x + threadIdx.x) * 2 + 1;
  if (idx1 < n)
    s_inScanData[2*threadIdx.x] = inData[idx1];
  if (idx2 < n)
    s_inScanData[2*threadIdx.x+1] = inData[idx2];
  __syncthreads();

  // 2. Each block does scan with data on SMEM
  // 2.1. Reduction phase
  for (int stride = 1; stride < 2 * blockDim.x; stride *= 2)
  {
    int s_idx = (threadIdx.x + 1) * 2 * stride - 1; // So active threads will be consecutive
    if (s_idx < 2 * blockDim.x)
      s_inScanData[s_idx] += s_inScanData[s_idx - stride];
    __syncthreads();
  }
  // 2.2. Post-reduction phase
  for (int stride = blockDim.x / 2; stride > 0; stride /= 2)
  {
    int s_idx = (threadIdx.x + 1) * 2 * stride - 1 + stride;
    if (s_idx < 2 * blockDim.x)
      s_inScanData[s_idx] += s_inScanData[s_idx - stride];
    __syncthreads();
  }

  // 3. Each block writes result from SMEM to GMEM
  // (each thread will write 2 elements)
  if (idx1 < n)
    outData[idx1] = s_inScanData[2*threadIdx.x];
  if (idx2 < n)
    outData[idx2] = s_inScanData[2*threadIdx.x+1];

  // 4. Each block write the result of the last thread block
  // (for regression if the array has large size)
  if (threadIdx.x == (blockDim.x-1) || (blockIdx.x * blockDim.x + threadIdx.x) == (n-1)/2)
    if (idx2 < n)
      outLastData[blockIdx.x] = s_inScanData[2 * threadIdx.x + 1];
    else
      outLastData[blockIdx.x] = s_inScanData[2 * threadIdx.x];
}

// Add scanned block sum to the before scanned block sum
//=====================================================================
__global__ void lastScan(unsigned int* odata, unsigned int* sums, const int n)
{
  if (blockIdx.x > 0)
  {
    int idx1 = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
    int idx2 = (blockIdx.x * blockDim.x + threadIdx.x) * 2 + 1;
    
    if (idx1 < n)
      odata[idx1] += sums[blockIdx.x - 1];
    if (idx2 < n)
      odata[idx2] += sums[blockIdx.x - 1];
  }
}

// Inclusive scan on large size array
//=====================================================================
void scanOnLargeSize(dim3 blockSize, dim3 gridSize, unsigned int *in, unsigned int *out, int n)
{
  dim3 _gridSize = gridSize;
  unsigned int *outlast;

  // Allocate device memory
  checkCudaErrors(hipMalloc((unsigned int**)&outlast, gridSize.x*sizeof(unsigned int)));
  
  // Launch scan kernel to find sum of each block (each thread block solve 2 value)
  inclusiveScan<<<gridSize, blockSize, 2*blockSize.x*sizeof(unsigned int)>>>(in, out, outlast, n);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  // Regress until gridSize just contain 1 block
  if (_gridSize.x > 1)
  {
    int nsize = _gridSize.x;
    unsigned int *sums;

    _gridSize.x = (_gridSize.x-1)/2/blockSize.x+1;
    checkCudaErrors(hipMalloc((unsigned int**)&sums, nsize*sizeof(unsigned int)));

    scanOnLargeSize(blockSize, _gridSize, outlast, sums, nsize);
      
    // Add scanned block sum (each thread block solve 2 value)
    lastScan<<<gridSize, blockSize>>>(out, sums, n);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    
    checkCudaErrors(hipFree(sums));
  }
  checkCudaErrors(hipFree(outlast));  
}

// Compute relative offset of each digit
//=====================================================================
__global__ void determineRelativeOffset(unsigned int* inData, unsigned int* exScanData,
					unsigned int* outData, int n)
{
  int idx = blockDim.x * blockIdx.x + threadIdx.x;

  if (idx < n)
  {
    int nZeros = n - exScanData[n - 1] - inData[n - 1]; // The number of zeros in array
    // Compute rank
    if (inData[idx] == 0)
      outData[idx] = idx - exScanData[idx];
    else
      outData[idx] = nZeros + exScanData[idx];
  }
}

// Scatter values corresponding to its location
//=====================================================================
__global__ void scatter(unsigned int* inData, unsigned int* outData,
		   	unsigned int* inDigit, unsigned int* exScanHist,
		   	unsigned int* offset, int n)
{
  int idx = blockDim.x * blockIdx.x + threadIdx.x;

  if (idx < n)
  {
    int location = exScanHist[inDigit[idx]] + offset[idx];
    outData[location] = inData[idx];
  }
}

//=====================================================================
void your_sort(unsigned int* const d_inputVals,
               unsigned int* const d_inputPos,
               unsigned int* const d_outputVals,
               unsigned int* const d_outputPos,
               const size_t numElems)
{ 
  //TODO
  //PUT YOUR SORT HERE
 
  // for takeCurrentDigit kernel
  const dim3 blockDigit(256);
  const dim3 gridDigit((numElems-1)/blockDigit.x+1);

  // for histogram kernel
  const dim3 blockHist(256);
  const dim3 gridHist((numElems-1)/blockHist.x+1);

  // for inclusive scan kernel
  const dim3 blockScan(512);
  const dim3 gridScan((numElems-1)/2/blockScan.x+1);

  // for compute offset kernel
  const dim3 blockOffset(256);
  const dim3 gridOffset((numElems-1)/blockOffset.x+1);

  // for scatter kernel
  const dim3 blockScatter(128);
  const dim3 gridScatter((numElems-1)/blockScatter.x+1);

  //
  unsigned int* d_vals_src = d_inputVals;
  unsigned int* d_pos_src = d_inputPos;
  unsigned int* d_vals_dst = d_outputVals;
  unsigned int* d_pos_dst = d_outputPos;

  //
  unsigned int nBits = 1;
  unsigned int nBins = 1 << nBits;

  //
  unsigned int *digit, *exScanDigit;
  unsigned int *hist;
  unsigned int *offset;

  checkCudaErrors(hipMalloc((unsigned int**)&digit, numElems*sizeof(unsigned int)));
  checkCudaErrors(hipMalloc((unsigned int**)&exScanDigit, (numElems+1)*sizeof(unsigned int)));
  checkCudaErrors(hipMalloc((unsigned int**)&hist, (nBins+1)*sizeof(unsigned int)));
  checkCudaErrors(hipMalloc((unsigned int**)&offset, numElems*sizeof(unsigned int)));

  // Loop from LSD to MSD
  // In each loop, soft values according to the current digit (using STABLE sort)
  for (unsigned int i = 0; i < 8*sizeof(unsigned int); i += nBits)
  {
    checkCudaErrors(hipMemset(hist, 0, (nBins+1)*sizeof(unsigned int)));
    checkCudaErrors(hipMemset(exScanDigit, 0, (numElems+1)*sizeof(unsigned int)));

    // Step 1: Compute histogram
    takeCurrentDigit<<<gridDigit, blockDigit>>>(d_vals_src, digit, numElems, i, nBins);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    histogram<<<gridHist, blockHist>>>(digit, hist+1, numElems);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    // Step 2: Compute relative offset
    scanOnLargeSize(blockScan, gridScan, digit, exScanDigit+1, numElems);

    determineRelativeOffset<<<gridOffset, blockOffset>>>(digit, exScanDigit, offset, numElems);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());	

    // Step 3: Scatter values to correct locations
    scatter<<<gridScatter, blockScatter>>>(d_vals_src, d_vals_dst, digit, hist, offset, numElems);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    scatter<<<gridScatter, blockScatter>>>(d_pos_src, d_pos_dst, digit, hist, offset, numElems);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    // Swap src and dst
    std::swap(d_vals_dst, d_vals_src);
    std::swap(d_pos_dst, d_pos_src);
  }
  
  // The above loop does an even number of iterations.
  // Its means the sorted array we need is store in input buffer,
  // so we need to copy from input buffer into output.
  checkCudaErrors(hipMemcpy(d_outputVals, d_inputVals, 
			numElems*sizeof(unsigned int), hipMemcpyDeviceToDevice));
  checkCudaErrors(hipMemcpy(d_outputPos, d_inputPos, 
			numElems*sizeof(unsigned int), hipMemcpyDeviceToDevice));

  // Free device memory
  checkCudaErrors(hipFree(digit));
  checkCudaErrors(hipFree(exScanDigit));
  checkCudaErrors(hipFree(hist));
  checkCudaErrors(hipFree(offset)); 
}
