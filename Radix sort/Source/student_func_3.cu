#include "hip/hip_runtime.h"
//Udacity HW 4
//Radix Sorting

#include "utils.h"

/* Red Eye Removal
   ===============
   
   For this assignment we are implementing red eye removal.  This is
   accomplished by first creating a score for every pixel that tells us how
   likely it is to be a red eye pixel.  We have already done this for you - you
   are receiving the scores and need to sort them in ascending order so that we
   know which pixels to alter to remove the red eye.

   Note: ascending order == smallest to largest

   Each score is associated with a position, when you sort the scores, you must
   also move the positions accordingly.

   Implementing Parallel Radix Sort with CUDA
   ==========================================

   The basic idea is to construct a histogram on each pass of how many of each
   "digit" there are.   Then we scan this histogram so that we know where to put
   the output of each digit.  For example, the first 1 must come after all the
   0s so we have to know how many 0s there are to be able to start moving 1s
   into the correct position.

   1) Histogram of the number of occurrences of each digit
   2) Exclusive Prefix Sum of Histogram
   3) Determine relative offset of each digit
        For example [0 0 1 1 0 0 1]
                ->  [0 1 0 1 2 3 2]
   4) Combine the results of steps 2 & 3 to determine the final
      output location for each element and move it there

   LSB Radix sort is an out-of-place sort and you will need to ping-pong values
   between the input and output buffers we have provided.  Make sure the final
   sorted results end up in the output buffer!  Hint: You may need to do a copy
   at the end.

 */
// Take current digit of each elements
//=====================================================================
__global__ void takeCurrentDigit(unsigned int* inData, unsigned int* outDigit,
			    	 int n, int currentDigit, int nBins)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < n)
  {
    outDigit[i] = (inData[i] >> currentDigit) & (nBins - 1);
  }  
}

// Histogram on digit
//=====================================================================
__global__ void sortAndHistogram(unsigned int* inData, unsigned int* inDigit,
			  unsigned int* hist, int n, int nBins)
{
  extern __shared__ unsigned int s_inDigit[];
  unsigned int* s_outData = &s_inDigit[blockDim.x];
  unsigned int* s_hist = &s_inDigit[2*blockDim.x];

  int idx = blockDim.x * blockIdx.x + threadIdx.x;

  // 1. Histogram
  // 1.1. Loads data and histogram on local data block on SMEM
  for (int i = threadIdx.x; i < nBins; i += blockDim.x)
    s_hist[i] = 0;
  __syncthreads();

  if (idx < n)
  {
    s_inDigit[threadIdx.x] = inDigit[idx];
    atomicAdd(&s_hist[s_inDigit[threadIdx.x]], 1);
  }
  __syncthreads();
  
  // 1.2. Loads histogram data from SMEM to GMEM
  for (int i = threadIdx.x; i < nBins; i += blockDim.x)
     hist[i*gridDim.x+blockIdx.x] = s_hist[i]; 
  
  // 2. Sort local data block on SMEM
  unsigned int location = 0;
  if (idx < n)
  {
    for (int i = 0; i < threadIdx.x; i++)
      location += (s_inDigit[i] <= s_inDigit[threadIdx.x]);
    int threshold = blockDim.x;
    if (blockIdx.x == gridDim.x - 1)
      threshold = n - blockDim.x * blockIdx.x;
    for (int i = threadIdx.x + 1; i < threshold; i++)
      location += (s_inDigit[i] < s_inDigit[threadIdx.x]);
    s_outData[location] = inData[idx];
  }
  __syncthreads();

  // 3. Loads sorted data from SMEM to GMEM
  if (idx < n)
  {
    inData[idx] = s_outData[threadIdx.x];
    inDigit[blockDim.x*blockIdx.x+location] = s_inDigit[threadIdx.x];
  }
}

// Inclusive scan
//=====================================================================
__global__ void inclusiveScan(unsigned int* inData, unsigned int* outData,
			      unsigned int* outLastData, int n)
{
  // 1. Each block loads data from GMEM to SMEM
  // (each thread will load 2 elements)
  extern __shared__ unsigned int s_inScanData[]; // It's size will be 2*blockDim.x (elements)
  int idx1 = (blockDim.x * blockIdx.x + threadIdx.x) * 2;
  int idx2 = (blockDim.x * blockIdx.x + threadIdx.x) * 2 + 1;
  if (idx1 < n)
    s_inScanData[2*threadIdx.x] = inData[idx1];
  if (idx2 < n)
    s_inScanData[2*threadIdx.x+1] = inData[idx2];
  __syncthreads();

  // 2. Each block does scan with data on SMEM
  // 2.1. Reduction phase
  for (int stride = 1; stride < 2 * blockDim.x; stride *= 2)
  {
    int s_idx = (threadIdx.x + 1) * 2 * stride - 1; // So active threads will be consecutive
    if (s_idx < 2 * blockDim.x)
      s_inScanData[s_idx] += s_inScanData[s_idx - stride];
    __syncthreads();
  }
  // 2.2. Post-reduction phase
  for (int stride = blockDim.x / 2; stride > 0; stride /= 2)
  {
    int s_idx = (threadIdx.x + 1) * 2 * stride - 1 + stride;
    if (s_idx < 2 * blockDim.x)
      s_inScanData[s_idx] += s_inScanData[s_idx - stride];
    __syncthreads();
  }

  // 3. Each block writes result from SMEM to GMEM
  // (each thread will write 2 elements)
  if (idx1 < n)
    outData[idx1] = s_inScanData[2*threadIdx.x];
  if (idx2 < n)
    outData[idx2] = s_inScanData[2*threadIdx.x+1];

  // 4. Each block write the result of the last thread block
  // (for regression if the array has large size)
  if (threadIdx.x == (blockDim.x-1) || (blockIdx.x * blockDim.x + threadIdx.x) == (n-1)/2)
    if (idx2 < n)
      outLastData[blockIdx.x] = s_inScanData[2 * threadIdx.x + 1];
    else
      outLastData[blockIdx.x] = s_inScanData[2 * threadIdx.x];
}

// Add scanned block sum to the before scanned block sum
//=====================================================================
__global__ void lastScan(unsigned int* odata, unsigned int* sums, const int n)
{
  if (blockIdx.x > 0)
  {
    int idx1 = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
    int idx2 = (blockIdx.x * blockDim.x + threadIdx.x) * 2 + 1;
    
    if (idx1 < n)
      odata[idx1] += sums[blockIdx.x - 1];
    if (idx2 < n)
      odata[idx2] += sums[blockIdx.x - 1];
  }
}

// Inclusive scan on large size array
//=====================================================================
void scanOnLargeSize(dim3 blockSize, dim3 gridSize, unsigned int *in, unsigned int *out, int n)
{
  dim3 _gridSize = gridSize;
  unsigned int *outlast;

  // Allocate device memory
  checkCudaErrors(hipMalloc((unsigned int**)&outlast, gridSize.x*sizeof(unsigned int)));
  
  // Launch scan kernel to find sum of each block (each thread block solve 2 value)
  inclusiveScan<<<gridSize, blockSize, 2*blockSize.x*sizeof(unsigned int)>>>(in, out, outlast, n);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  // Regress until gridSize just contain 1 block
  if (_gridSize.x > 1)
  {
    int nsize = _gridSize.x;
    unsigned int *sums;

    _gridSize.x = (_gridSize.x-1)/2/blockSize.x+1;
    checkCudaErrors(hipMalloc((unsigned int**)&sums, nsize*sizeof(unsigned int)));

    scanOnLargeSize(blockSize, _gridSize, outlast, sums, nsize);
      
    // Add scanned block sum (each thread block solve 2 value)
    lastScan<<<gridSize, blockSize>>>(out, sums, n);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    
    checkCudaErrors(hipFree(sums));
  }
  checkCudaErrors(hipFree(outlast));  
}

// Compute rank of each nBits-digit
//=====================================================================
__global__ void computeRank(unsigned int* inData, unsigned int* outData, int n)
{
  extern __shared__ unsigned int s_inData[];
  int idx = blockDim.x * blockIdx.x + threadIdx.x;

  if (idx < n)
    s_inData[threadIdx.x] = inData[idx];
  __syncthreads();

  if (idx < n)
  {
    // Compute rank on local block
    unsigned int rank = 0;
    for (int i = threadIdx.x - 1; i >= 0; i--)
    {
      if (s_inData[i] < s_inData[threadIdx.x])
	break;
      rank++;
    }
    outData[idx] = rank;
  }
}

// Scatter values corresponding to its location
//=====================================================================
__global__ void scatter(unsigned int* inData1, unsigned int* outData1,
			unsigned int* inData2, unsigned int* outData2,
		   	unsigned int* inDigit, unsigned int* exScanHist,
		    	unsigned int* rank, int n)
{
  int idx = blockDim.x * blockIdx.x + threadIdx.x;

  if (idx < n)
  {
    outData1[exScanHist[inDigit[idx]*gridDim.x+blockIdx.x] + rank[idx]] = inData1[idx];
    outData2[exScanHist[inDigit[idx]*gridDim.x+blockIdx.x] + rank[idx]] = inData2[idx];
  }
}

//=====================================================================
void your_sort(unsigned int* const d_inputVals,
               unsigned int* const d_inputPos,
               unsigned int* const d_outputVals,
               unsigned int* const d_outputPos,
               const size_t numElems)
{ 
  //TODO
  //PUT YOUR SORT HERE
  
  //
  unsigned int nBits = 8;
  unsigned int nBins = 1 << nBits;
 
  // for takeCurrentDigit kernel
  const dim3 blockDigit(256);
  const dim3 gridDigit((numElems-1)/blockDigit.x+1);

  // for histogram kernel, compute rank kernel and scatter kernel
  const dim3 blockSize(128);
  const dim3 gridSize((numElems-1)/blockSize.x+1);

  // for inclusive scan kernel
  const dim3 blockScan(512);
  const dim3 gridScan((gridSize.x*nBins-1)/2/blockScan.x+1);

  //
  unsigned int* d_vals_src = d_inputVals;
  unsigned int* d_pos_src = d_inputPos;
  unsigned int* d_vals_dst = d_outputVals;
  unsigned int* d_pos_dst = d_outputPos;

  //
  unsigned int *digit;
  unsigned int *hist, *exScanHist;
  unsigned int *rank;
  
  checkCudaErrors(hipMalloc((unsigned int**)&digit, numElems*sizeof(unsigned int)));
  checkCudaErrors(hipMalloc((unsigned int**)&hist, 
				nBins*gridSize.x*sizeof(unsigned int)));
  checkCudaErrors(hipMalloc((unsigned int**)&exScanHist, 
				(nBins*gridSize.x+1)*sizeof(unsigned int)));
  checkCudaErrors(hipMalloc((unsigned int**)&rank, numElems*sizeof(unsigned int)));

  // Loop from LSD to MSD
  // In each loop, soft values according to the current digit (using STABLE sort)
  for (unsigned int i = 0; i < 8*sizeof(unsigned int); i += nBits)
  {
    checkCudaErrors(hipMemset(hist, 0, nBins*gridSize.x*sizeof(unsigned int)));
    checkCudaErrors(hipMemset(exScanHist, 0, (nBins*gridSize.x+1)*sizeof(unsigned int)));

    // Step 1: Compute histogram
    takeCurrentDigit<<<gridDigit, blockDigit>>>(d_vals_src, digit, numElems, i, nBins);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    sortAndHistogram<<<gridSize, blockSize, (2*blockSize.x+nBins)*sizeof(unsigned int)>>>
							(d_vals_src, digit, hist, numElems, nBins);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    // Step 2: Scan on histogram
    scanOnLargeSize(blockScan, gridScan, hist, exScanHist+1, nBins*gridSize.x);

    // Step 3: Compute rank of each nBits-digit
    computeRank<<<gridSize, blockSize, blockSize.x*sizeof(unsigned int)>>>(digit, rank, numElems);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    // Step 4: Scatter values on correct location    
    scatter<<<gridSize, blockSize>>>(d_vals_src, d_vals_dst, d_pos_src, d_pos_dst,
				     digit, exScanHist, rank, numElems);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    // Swap src and dst
    std::swap(d_vals_dst, d_vals_src);
    std::swap(d_pos_dst, d_pos_src);
  }
  
  // The above loop does an even number of iterations.
  // Its means the sorted array we need is store in input buffer,
  // so we need to copy from input buffer into output.
  checkCudaErrors(hipMemcpy(d_outputVals, d_inputVals, 
			numElems*sizeof(unsigned int), hipMemcpyDeviceToDevice));
  checkCudaErrors(hipMemcpy(d_outputPos, d_inputPos, 
			numElems*sizeof(unsigned int), hipMemcpyDeviceToDevice));

  // Free device memory
  checkCudaErrors(hipFree(digit));
  checkCudaErrors(hipFree(hist));
  checkCudaErrors(hipFree(exScanHist));
  checkCudaErrors(hipFree(rank));
}
